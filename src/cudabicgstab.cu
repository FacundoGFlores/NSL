#include "hip/hip_runtime.h"
/*****************************************************************************
Name		: 	bicgstab.cpp
Author      : 	Flores, Facundo Gabriel
e-mail		:   flores.facundogabriel@gmail.com
Version     : 	0.1
Description : 	This is implementation of the bi-conjugate gradient stable
				using cuBlas library
License     :
    			Copyright (C) 2012 Flores, Facundo Gabriel

    			This program is free software: you can redistribute it and/or modify
    			it under the terms of the GNU General Public License as published by
    			the Free Software Foundation, either version 3 of the License, or
    			(at your option) any later version.

    			This program is distributed in the hope that it will be useful,
    			but WITHOUT ANY WARRANTY; without even the implied warranty of
    			MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    			GNU General Public License for more details.

    			You should have received a copy of the GNU General Public License
    			along with this program.  If not, see <http://www.gnu.org/licenses/>.

**************************************************************************** */

#include <iostream>

#include <cstdlib>

#include <hip/hip_runtime.h>

#include <hipblas.h>

#include <vector>

#include <sys/time.h>

#include <cudabicgstab.cuh>

using namespace std;

void show(float *vec, int N)
{
	for(int i = 0; i < N; i++)
		cout << vec[i] << endl;
}

/**
 * Take the vector from the linear container
 * @param dst the C++ vector
 * @param vec the C array
 */
void cudabicgstabGetArray(float *dst, const vector<float> &vec)
{
	for(int i = 0; i < vec.size(); i++)
		dst[i] = vec[i];
}

void cudabicgstabFillZeros(float *dst, const int N)
{
	for(int i = 0; i < N; i++)
		dst[i] = 0;
}

/**
 * Solve a linear system of equations using the
 * bi-conjugate gradient stable method.
 * @param Matrix_A Matrix A
 * @param Vector_B Vector B
 * @param Vector_X Vector X
 * @param e allowed error
 * @param MaxIter allowed iterations
 * @return the time spent
 */
long cudaBicgstab(const linearMatrix &Matrix_A, const linearMatrix &Vector_B, linearMatrix &Vector_X,
						const float e, const int MaxIter)
{

	/* ********** ALGORITHM HOST VARIABLES ********** */

	int N = Vector_B.Get_Width();

	float *mA = (float *)malloc(sizeof(float) * N * N);
	if(!mA)
	{
		cout << "Cannot allocate memory" << endl;
		exit(EXIT_FAILURE);
	}

	float *vB = (float *)malloc(sizeof(float) * N);
	if(!vB)
	{
		cout << "Cannot allocate memory" << endl;
		exit(EXIT_FAILURE);
	}

	float *vX = (float *)malloc(sizeof(float) * N);
	if(!vX)
	{
		cout << "Cannot allocate memory" << endl;
		exit(EXIT_FAILURE);
	}

	float *nullvec = (float *)malloc(sizeof(float) * N);
	if(!nullvec)
	{
		cout << "Cannot allocate memory" << endl;
		exit(EXIT_FAILURE);
	}

	//Set vectors
	cudabicgstabGetArray(mA, Matrix_A.Get_Vector());
	cudabicgstabGetArray(vB, Vector_B.Get_Vector());
	cudabicgstabGetArray(vX, Vector_X.Get_Vector());

	cudabicgstabFillZeros(nullvec, N);
	/* ************************************ */

	/* ********** DEVICE VARIABLES ********** */

	hipError_t cudaStat ;
	hipblasStatus_t stat ;
	hipblasHandle_t handle ;

	float *dev_A;
	float *dev_B;
	float *dev_X;
	float *dev_tmp;
	float *dev_tmp1;
	float *dev_tmp2;
	float *dev_r;
	float *dev_rhat0;
	float *dev_v;
	float *dev_p;
	float *dev_s;
	float *dev_t;
	float *dev_r_tmp;
	float *dev_null;

	/* ======================== Allocating ======================== */
	cudaStat = hipMalloc( (void **)& dev_null, N * sizeof(float) );
	if(cudaStat != EXIT_SUCCESS)
	{
		cout << "device memory allocation failed" << endl;
		exit(EXIT_FAILURE);
	}

	cudaStat = hipMalloc( (void **)& dev_A, N * N * sizeof(float) );
	if(cudaStat != EXIT_SUCCESS)
	{
		cout << "device memory allocation failed" << endl;
		exit(EXIT_FAILURE);
	}

	cudaStat = hipMalloc( (void **)& dev_B, N * sizeof(float) );
	if(cudaStat != EXIT_SUCCESS)
	{
		cout << "device memory allocation failed" << endl;;
		exit(EXIT_FAILURE);
	}

	cudaStat = hipMalloc( (void **)& dev_X, N * sizeof(float) );
	if(cudaStat != EXIT_SUCCESS)
	{
		cout << "device memory allocation failed" << endl;
		exit(EXIT_FAILURE);
	}

	cudaStat = hipMalloc( (void **)& dev_tmp, N * sizeof(float) );
	if(cudaStat != EXIT_SUCCESS)
	{
		cout << "device memory allocation failed" << endl;
		exit(EXIT_FAILURE);
	}

	cudaStat = hipMalloc( (void **)& dev_tmp1, N * sizeof(float) );
	if(cudaStat != EXIT_SUCCESS)
	{
		cout << "device memory allocation failed" << endl;
		exit(EXIT_FAILURE);
	}

	cudaStat = hipMalloc( (void **)& dev_tmp2, N * sizeof(float) );
	if(cudaStat != EXIT_SUCCESS)
	{
		cout << "device memory allocation failed" << endl;
		exit(EXIT_FAILURE);
	}

	cudaStat = hipMalloc( (void **)& dev_r, N * sizeof(float) );
	if(cudaStat != EXIT_SUCCESS)
	{
		cout << "device memory allocation failed" << endl;
		exit(EXIT_FAILURE);
	}

	cudaStat = hipMalloc( (void **)& dev_rhat0, N * sizeof(float) );
	if(cudaStat != EXIT_SUCCESS)
	{
		cout << "device memory allocation failed" << endl;
		exit(EXIT_FAILURE);
	}

	cudaStat = hipMalloc( (void **)& dev_v, N * sizeof(float) );
	if(cudaStat != EXIT_SUCCESS)
	{
		cout << "device memory allocation failed" << endl;
		exit(EXIT_FAILURE);
	}

	cudaStat = hipMalloc( (void **)& dev_p, N * sizeof(float) );
	if(cudaStat != EXIT_SUCCESS)
	{
		cout << "device memory allocation failed" << endl;
		exit(EXIT_FAILURE);
	}

	cudaStat = hipMalloc( (void **)& dev_s, N * sizeof(float) );
	if(cudaStat != EXIT_SUCCESS)
	{
		cout << "device memory allocation failed" << endl;
		exit(EXIT_FAILURE);
	}

	cudaStat = hipMalloc( (void **)& dev_r_tmp, N * sizeof(float) );
	if(cudaStat != EXIT_SUCCESS)
	{
		cout << "device memory allocation failed" << endl;
		exit(EXIT_FAILURE);
	}

	cudaStat = hipMalloc( (void **)& dev_t, N * sizeof(float) );
	if(cudaStat != EXIT_SUCCESS)
	{
		cout << "device memory allocation failed" << endl;
		exit(EXIT_FAILURE);
	}
	/* ================================================================ */

	/* ==================== Settings ==================== */
	stat = hipblasCreate(&handle);
	if(stat != HIPBLAS_STATUS_SUCCESS)
	{
		cout << "CUBLAS Initialization failed!" << endl;
		exit(EXIT_FAILURE);
	}

	stat = hipblasSetMatrix(N, N, sizeof(float), mA, N, dev_A, N);
	if(stat != HIPBLAS_STATUS_SUCCESS)
	{
		cout << "CUBLAS setting matrix failed" << endl;
		exit(EXIT_FAILURE);
	}

	stat = hipblasSetVector(N, sizeof(float), nullvec, 1, dev_null, 1);
	if(stat != HIPBLAS_STATUS_SUCCESS)
	{
		cout << "CUBLAS setting vectorB failed" << endl;
		exit(EXIT_FAILURE);
	}

	stat = hipblasSetVector(N, sizeof(float), vB, 1, dev_B, 1);
	if(stat != HIPBLAS_STATUS_SUCCESS)
	{
		cout << "CUBLAS setting vectorB failed" << endl;
		exit(EXIT_FAILURE);
	}

	stat = hipblasSetVector(N, sizeof(float), nullvec, 1, dev_X, 1);
	if(stat != HIPBLAS_STATUS_SUCCESS)
	{
		cout << "CUBLAS setting vectorX failed" << endl;
		exit(EXIT_FAILURE);
	}

	stat = hipblasSetVector(N, sizeof(float), nullvec, 1, dev_tmp, 1);
	if(stat != HIPBLAS_STATUS_SUCCESS)
	{
		cout << "CUBLAS setting vector tmp failed" << endl;
		exit(EXIT_FAILURE);
	}

	stat = hipblasSetVector(N, sizeof(float), nullvec, 1, dev_tmp1, 1);
	if(stat != HIPBLAS_STATUS_SUCCESS)
	{
		cout << "CUBLAS setting vector tmp1 failed" << endl;
		exit(EXIT_FAILURE);
	}

	stat = hipblasSetVector(N, sizeof(float), nullvec, 1, dev_tmp2, 1);
	if(stat != HIPBLAS_STATUS_SUCCESS)
	{
		cout << "CUBLAS setting vector tmp2 failed" << endl;
		exit(EXIT_FAILURE);
	}

	stat = hipblasSetVector(N, sizeof(float), nullvec, 1, dev_r, 1);
	if(stat != HIPBLAS_STATUS_SUCCESS)
	{
		cout << "CUBLAS setting vector r failed" << endl;
		exit(EXIT_FAILURE);
	}

	stat = hipblasSetVector(N, sizeof(float), nullvec, 1, dev_rhat0, 1);
	if(stat != HIPBLAS_STATUS_SUCCESS)
	{
		cout << "CUBLAS setting vector r failed" << endl;
		exit(EXIT_FAILURE);
	}

	stat = hipblasSetVector(N, sizeof(float), nullvec, 1, dev_v, 1);
	if(stat != HIPBLAS_STATUS_SUCCESS)
	{
		cout << "CUBLAS setting vector v failed" << endl;
		exit(EXIT_FAILURE);
	}

	stat = hipblasSetVector(N, sizeof(float), nullvec, 1, dev_p, 1);
	if(stat != HIPBLAS_STATUS_SUCCESS)
	{
		cout << "CUBLAS setting vector p failed" << endl;
		exit(EXIT_FAILURE);
	}

	stat = hipblasSetVector(N, sizeof(float), nullvec, 1, dev_s, 1);
	if(stat != HIPBLAS_STATUS_SUCCESS)
	{
		cout << "CUBLAS setting vector s failed" << endl;
		exit(EXIT_FAILURE);
	}

	stat = hipblasSetVector(N, sizeof(float), nullvec, 1, dev_r_tmp, 1);
	if(stat != HIPBLAS_STATUS_SUCCESS)
	{
		cout << "CUBLAS setting vector r tmp failed" << endl;
		exit(EXIT_FAILURE);
	}

	stat = hipblasSetVector(N, sizeof(float), nullvec, 1, dev_t, 1);
	if(stat != HIPBLAS_STATUS_SUCCESS)
	{
		cout << "CUBLAS setting vector t failed" << endl;
		exit(EXIT_FAILURE);
	}

	/* ======================================================== */
	/* ************************************** */

	/* ********** BICGSTAB Method ********** */


    timeval start;
    timeval end;

    long seconds, useconds, final;

    gettimeofday(&start, 0);

	float gemv_alpha = -1;
	float gemv_betha = 1;

	//r = b - Ax
	hipblasSgemv(handle, HIPBLAS_OP_N, N, N, &gemv_alpha, dev_A, N, dev_X, 1, &gemv_betha, dev_tmp, 1);
	gemv_alpha = 1;
	hipblasSaxpy(handle, N, &gemv_alpha, dev_B, 1, dev_tmp, 1);
	hipMemcpy(dev_r, dev_tmp, N * sizeof(float), hipMemcpyDeviceToDevice);
	hipMemcpy(dev_rhat0, dev_r, N * sizeof(float), hipMemcpyDeviceToDevice);

	float betha;
	float alpha = 1;
	float w = 1;
	float rho_old = 1;
	float aux_norm;
	float dotprod;
	float lasterror;

	int k = 0;

	//rho = <r,r>
	hipblasSdot(handle, N, dev_r, 1, dev_r, 1, &dotprod);
	float rho = dotprod;
	lasterror = sqrt(rho);

	//||Vector_B||
	hipblasSdot(handle, N, dev_B, 1, dev_B, 1, &dotprod);
	float norm_b = sqrt(dotprod);
	float new_e = e * norm_b;



	while(lasterror > new_e  && k < MaxIter)
	{
		k++;

		betha = (rho / rho_old) * (alpha / w);

		//p = r + betha(p - w*v)
		gemv_betha = (-1) * w;
		hipMemcpy(dev_tmp, dev_p, N * sizeof(float), hipMemcpyDeviceToDevice);
		hipblasSaxpy(handle, N, &gemv_betha, dev_v, 1, dev_tmp, 1);
		hipMemcpy(dev_tmp1, dev_r, N * sizeof(float), hipMemcpyDeviceToDevice);
		gemv_betha = betha;
		hipblasSaxpy(handle, N, &gemv_betha, dev_tmp, 1, dev_tmp1, 1);
		hipMemcpy(dev_p, dev_tmp1, N * sizeof(float), hipMemcpyDeviceToDevice);

		//v = A*p
		gemv_alpha = 1;
		gemv_betha = 1;
		hipMemcpy(dev_tmp, dev_null, N * sizeof(float), hipMemcpyDeviceToDevice);
		hipblasSgemv(handle, HIPBLAS_OP_N, N, N, &gemv_alpha, dev_A, N, dev_p, 1, &gemv_betha, dev_tmp, 1);
		hipMemcpy(dev_v, dev_tmp, N * sizeof(float), hipMemcpyDeviceToDevice);

		//alpha = rho / <rhat0,v>
		hipblasSdot(handle, N, dev_rhat0, 1, dev_v, 1, &dotprod);
		alpha = rho / dotprod;

		//s = r - alpha * v
		gemv_alpha = (-1) * alpha;
		hipMemcpy(dev_tmp, dev_r, N * sizeof(float), hipMemcpyDeviceToDevice);
		hipblasSaxpy(handle, N, &gemv_alpha, dev_v, 1, dev_tmp, 1);
		hipMemcpy(dev_s, dev_tmp, N * sizeof(float), hipMemcpyDeviceToDevice);

		//t = A * s
		gemv_alpha = 1;
		gemv_betha = 1;
		hipMemcpy(dev_tmp, dev_null, N * sizeof(float), hipMemcpyDeviceToDevice);
		hipblasSgemv(handle, HIPBLAS_OP_N, N, N, &gemv_alpha, dev_A, N, dev_s, 1, &gemv_betha, dev_tmp, 1);
		hipMemcpy(dev_t, dev_tmp, N * sizeof(float), hipMemcpyDeviceToDevice);

		//w = <t,s>/sqrt(<t,t>)
		hipblasSdot(handle, N, dev_t, 1, dev_t, 1, &dotprod);
		aux_norm = sqrt(dotprod);
		hipblasSdot(handle, N, dev_t, 1, dev_s, 1, &dotprod);
		w = dotprod / aux_norm;

		//rho = -w*rhat0*t
		rho_old = rho;
		hipblasSdot(handle, N, dev_rhat0, 1, dev_t, 1, &dotprod);
		rho = dotprod;
		rho = rho * (-1) * w;

		//x + alpha * p + w * s
		hipMemcpy(dev_tmp1, dev_X, N * sizeof(float), hipMemcpyDeviceToDevice);
		gemv_alpha = alpha;
		hipblasSaxpy(handle, N, &gemv_alpha, dev_p, 1, dev_tmp1, 1);
		hipMemcpy(dev_tmp2, dev_tmp1, N * sizeof(float), hipMemcpyDeviceToDevice);
		gemv_alpha = w;
		hipblasSaxpy(handle, N, &gemv_alpha, dev_s, 1, dev_tmp2, 1);
		hipMemcpy(dev_X, dev_tmp2, N * sizeof(float), hipMemcpyDeviceToDevice);

		//r = s- w * t
		hipMemcpy(dev_tmp, dev_s, N * sizeof(float), hipMemcpyDeviceToDevice);
		gemv_alpha = (-1) * w;
		hipblasSaxpy(handle, N, &gemv_alpha, dev_t, 1, dev_tmp, 1);
		hipMemcpy(dev_r, dev_tmp, N * sizeof(float), hipMemcpyDeviceToDevice);

		//lasterror = sqrt(<r,r>)
		hipblasSdot(handle, N, dev_r, 1, dev_r, 1, &dotprod);
		lasterror = sqrt(dotprod);
	}
	gettimeofday(&end, 0);

	seconds  = end.tv_sec  - start.tv_sec;

	useconds = end.tv_usec - start.tv_usec;

	final = ((seconds) * 1000 + useconds/1000.0) + 0.5;

	if(k == MaxIter)
		cout << "Can't Solve that system of equations!" << endl;

	//Give the result to host
	hipMemcpy(vX, dev_X, N * sizeof(float), hipMemcpyDeviceToHost);
	vector<float> v_tmp(vX, vX + N);
	Vector_X.Set_Matrix(v_tmp, N, 1);

	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_X);
	hipFree(dev_tmp);
	hipFree(dev_tmp1);
	hipFree(dev_tmp2);
	hipFree(dev_r);
	hipFree(dev_rhat0);
	hipFree(dev_v);
	hipFree(dev_p);
	hipFree(dev_s);
	hipFree(dev_t);
	hipFree(dev_r_tmp);
	hipFree(dev_null);



	free(mA);
	free(vB);
	free(vX);
	free(nullvec);

	return final;
}

